#include "hip/hip_runtime.h"
#include <algorithm>
#include <string>
#define _USE_MATH_DEFINES
#include <cfloat>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

#define BLOCK_WIDTH 32
#define THREADS_PER_BLOCK (BLOCK_WIDTH * BLOCK_WIDTH)
#define BATCH_SIZE (THREADS_PER_BLOCK * 2)

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
#else
#define cudaCheckError(ans) ans
#endif

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants {

  SceneName sceneName;

  int numTriangles;
  float *vertices;
  float *colors;

  int imageWidth;
  int imageHeight;
  float *imageData;

  float combinedMatrix[4][4];
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// Read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int cuConstNoiseYPermutationTable[256];
__constant__ int cuConstNoiseXPermutationTable[256];
__constant__ float cuConstNoise1DValueTable[256];

// Color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float cuConstColorRamp[COLOR_MAP_SIZE][3];

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

  int imageX = blockIdx.x * blockDim.x + threadIdx.x;
  int imageY = blockIdx.y * blockDim.y + threadIdx.y;

  int width = cuConstRendererParams.imageWidth;
  int height = cuConstRendererParams.imageHeight;

  if (imageX >= width || imageY >= height)
    return;

  int offset = 4 * (imageY * width + imageX);
  float4 value = make_float4(r, g, b, a);

  // Write to global memory: As an optimization, this code uses a float4
  // store, which results in more efficient code than if it were coded as
  // four separate float stores.
  *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceCamera -- (CUDA device code)
//
// Update the camera position for the camera path animation
__global__ void kernelAdvanceCamera() {}

// scan kernel that is only valid for N <= 2 * THREADS_PER_BLOCK
// use template to unroll the for loop in compile time
template <int N> __device__ void scanKernel(short *data) {
  int threadId = threadIdx.y * blockDim.x + threadIdx.x;
// upsweep phase.
#pragma unroll
  for (int twod = 1; twod < N; twod *= 2) {
    int twod1 = twod * 2;
    for (int i = threadId * twod1; i < N; i += THREADS_PER_BLOCK * twod1) {
      data[i + twod1 - 1] += data[i + twod - 1];
    }
    __syncthreads();
  }
  if (threadId == 0) {
    data[N - 1] = 0;
  }
  __syncthreads();

// downsweep phase.
#pragma unroll
  for (int twod = N / 2; twod >= 1; twod /= 2) {
    int twod1 = twod * 2;
    for (int i = threadId * twod1; i < N; i += THREADS_PER_BLOCK * twod1) {
      int t = data[i + twod - 1];
      data[i + twod - 1] = data[i + twod1 - 1];
      // change twod1 below to twod to reverse prefix sum.
      data[i + twod1 - 1] += t;
    }
    __syncthreads();
  }
}

// scan kernel that is only valid for N <= 2 * THREADS_PER_BLOCK
__device__ void scanKernelVariable(short *data, int N) {
  int threadId = threadIdx.y * blockDim.x + threadIdx.x;
  // upsweep phase.
  for (int twod = 1; twod < N; twod *= 2) {
    int twod1 = twod * 2;
    for (int i = threadId * twod1; i < N; i += THREADS_PER_BLOCK * twod1) {
      data[i + twod1 - 1] += data[i + twod - 1];
    }
    __syncthreads();
  }
  if (threadId == 0) {
    data[N - 1] = 0;
  }
  __syncthreads();

  // downsweep phase.
  for (int twod = N / 2; twod >= 1; twod /= 2) {
    int twod1 = twod * 2;
    for (int i = threadId * twod1; i < N; i += THREADS_PER_BLOCK * twod1) {
      int t = data[i + twod - 1];
      data[i + twod - 1] = data[i + twod1 - 1];
      // change twod1 below to twod to reverse prefix sum.
      data[i + twod1 - 1] += t;
    }
    __syncthreads();
  }
}

// write index in input to output
__device__ void writeIndex(short *input, short *output, int N) {
  int threadId = threadIdx.y * blockDim.x + threadIdx.x;

  for (int offset = 0; offset < N; offset += THREADS_PER_BLOCK) {
    if (offset + threadId < N && offset + threadId > 0) {
      int curr = input[offset + threadId];
      if (curr > input[offset + threadId - 1]) {
        output[curr - 1] = offset + threadId - 1;
      }
    }

    __syncthreads();
  }
}

/* Helper function to round up to a power of 2.
 */
__device__ inline int nextPow2(int n) {
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  n++;
  return n;
}

// check if a circle intersects with box
__device__ __inline__ int circleInBox(float circleX, float circleY,
                                      float circleRadius, float boxL,
                                      float boxR, float boxT, float boxB) {

  // clamp circle center to box (finds the closest point on the box)
  float closestX =
      (circleX > boxL) ? ((circleX < boxR) ? circleX : boxR) : boxL;
  float closestY =
      (circleY > boxB) ? ((circleY < boxT) ? circleY : boxT) : boxB;

  // is circle radius less than the distance to the closest point on
  // the box?
  float distX = closestX - circleX;
  float distY = closestY - circleY;

  if (((distX * distX) + (distY * distY)) <= (circleRadius * circleRadius)) {
    return 1;
  } else {
    return 0;
  }
}

static __device__ void transformVertex(float *vertex, float *projectedVertex,
                                       float combinedMatrix[4][4]) {
  float vec4[4] = {vertex[0], vertex[1], vertex[2], 1.0f};

  float clipSpaceVertex[4];
  for (int i = 0; i < 4; ++i) {
    clipSpaceVertex[i] =
        combinedMatrix[i][0] * vec4[0] + combinedMatrix[i][1] * vec4[1] +
        combinedMatrix[i][2] * vec4[2] + combinedMatrix[i][3] * vec4[3];
  }
  projectedVertex[0] = (clipSpaceVertex[0] / clipSpaceVertex[3] + 1.0) / 2.0;
  projectedVertex[1] = (clipSpaceVertex[1] / clipSpaceVertex[3] + 1.0) / 2.0;
  projectedVertex[2] = clipSpaceVertex[2] / clipSpaceVertex[3];
}

static __device__ float getTriangleZ(float px, float py,
                                     float *projectedVertices) {
  float x1 = projectedVertices[0];
  float y1 = projectedVertices[1];
  float x2 = projectedVertices[3];
  float y2 = projectedVertices[4];
  float x3 = projectedVertices[6];
  float y3 = projectedVertices[7];
  float lambda1 = ((y2 - y3) * (px - x3) + (x3 - x2) * (py - y3)) /
                  ((y2 - y3) * (x1 - x3) + (x3 - x2) * (y1 - y3));
  float lambda2 = ((y3 - y1) * (px - x3) + (x1 - x3) * (py - y3)) /
                  ((y2 - y3) * (x1 - x3) + (x3 - x2) * (y1 - y3));
  float lambda3 = 1 - lambda1 - lambda2;
  if (lambda1 >= 0 && lambda2 >= 0 && lambda3 >= 0 && lambda1 <= 1 &&
      lambda2 <= 1 && lambda3 <= 1) {
    float result = lambda1 * projectedVertices[2] +
                   lambda2 * projectedVertices[5] +
                   lambda3 * projectedVertices[8];
    return result;
  } else {
    return -1;
  }
}

static __device__ void rasterization(int numTriangles, float *projectedVertices,
                                     const float *vertices, const float *colors,
                                     float *outColor, float x, float y) {
  float minZ = FLT_MAX;
  for (int i = 0; i < numTriangles; i++) {
    float z = getTriangleZ(x, y, projectedVertices + i * 9);
    if (z < 0) {
      // not in triangle
      continue;
    }
    if (z < minZ) {
      minZ = z;
      outColor[0] = colors[i * 4];
      outColor[1] = colors[i * 4 + 1];
      outColor[2] = colors[i * 4 + 2];
      outColor[3] = colors[i * 4 + 3];
    }
  }
}

__global__ void kernelProjectVertices(float *combinedMatrix,
                                      float *projectedVertices) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float sCombinedMatrix[4][4];

  if (threadIdx.x < 16) {
    sCombinedMatrix[threadIdx.x / 4][threadIdx.x % 4] =
        combinedMatrix[threadIdx.x];
  }

  // project vertices
  if (idx < cuConstRendererParams.numTriangles * 3) {
    // printf("original vertices: %f %f %f\n",
    //  cuConstRendererParams.vertices[3 * idx],
    //  cuConstRendererParams.vertices[3 * idx + 1],
    //  cuConstRendererParams.vertices[3 * idx + 2]);
    transformVertex(&cuConstRendererParams.vertices[3 * idx],
                    projectedVertices + idx * 3, sCombinedMatrix);
    // printf("projectedVertices: %f %f %f\n", projectedVertices[idx * 3],
    //        projectedVertices[idx * 3 + 1], projectedVertices[idx * 3 + 2]);
  }
}

// render the image by pixels
__global__ void kernelRenderPixels(float *projectedVertices) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // rasterization
  float px = (x + 0.5f) / cuConstRendererParams.imageWidth;
  float py = (y + 0.5f) / cuConstRendererParams.imageHeight;
  rasterization(cuConstRendererParams.numTriangles, projectedVertices,
                cuConstRendererParams.vertices, cuConstRendererParams.colors,
                &cuConstRendererParams
                     .imageData[4 * (y * cuConstRendererParams.imageWidth + x)],
                px, py);
}

////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
  image = NULL;

  cudaDeviceVertices = NULL;
  cudaDeviceEndIndices = NULL;
  cudaDeviceColors = NULL;
}

CudaRenderer::~CudaRenderer() {

  if (image) {
    delete image;
  }

  if (cudaDeviceVertices) {
    hipFree(cudaDeviceVertices);
    hipFree(cudaDeviceEndIndices);
    hipFree(cudaDeviceColors);
  }
}

const Image *CudaRenderer::getImage() {

  // Need to copy contents of the rendered image from device memory
  // before we expose the Image object to the caller

  printf("Copying image data from device\n");

  hipMemcpy(image->data, cudaDeviceImageData,
             sizeof(float) * 4 * image->width * image->height,
             hipMemcpyDeviceToHost);

  return image;
}

void CudaRenderer::loadScene(SceneName name) {
  sceneName = name;
  scene = ::loadScene(sceneName, image->width, image->height);
  scene->serialize(numTriangles, vertices, colors);
}

void CudaRenderer::setup() {

  int deviceCount = 0;
  bool isFastGPU = false;
  std::string name;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Initializing CUDA for CudaRenderer\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    name = deviceProps.name;
    if (name.compare("GeForce RTX 2080") == 0) {
      isFastGPU = true;
    }

    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
  if (!isFastGPU) {
    printf("WARNING: "
           "You're not running on a fast GPU, please consider using "
           "NVIDIA RTX 2080.\n");
    printf("---------------------------------------------------------\n");
  }

  // By this time the scene should be loaded.  Now copy all the key
  // data structures into device memory so they are accessible to
  // CUDA kernels
  //
  // See the CUDA Programmer's Guide for descriptions of
  // hipMalloc and hipMemcpy

  hipMalloc(&cudaDeviceVertices, sizeof(float) * 9 * numTriangles);
  hipMalloc(&cudaDeviceColors, sizeof(float) * 4 * numTriangles);
  hipMalloc(&cudaDeviceImageData,
             sizeof(float) * 4 * image->width * image->height);

  hipMemcpy(cudaDeviceVertices, vertices, sizeof(float) * 9 * numTriangles,
             hipMemcpyHostToDevice);
  hipMemcpy(cudaDeviceColors, colors, sizeof(float) * 4 * numTriangles,
             hipMemcpyHostToDevice);

  // Initialize parameters in constant memory.  We didn't talk about
  // constant memory in class, but the use of read-only constant
  // memory here is an optimization over just sticking these values
  // in device global memory.  NVIDIA GPUs have a few special tricks
  // for optimizing access to constant memory.  Using global memory
  // here would have worked just as well.  See the Programmer's
  // Guide for more information about constant memory.

  GlobalConstants params;
  params.sceneName = sceneName;
  params.numTriangles = numTriangles;
  params.vertices = cudaDeviceVertices;
  params.colors = cudaDeviceColors;
  params.imageWidth = image->width;
  params.imageHeight = image->height;
  params.imageData = cudaDeviceImageData;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

  // Also need to copy over the noise lookup tables, so we can
  // implement noise on the GPU
  int *permX;
  int *permY;
  float *value1D;
  getNoiseTables(&permX, &permY, &value1D);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
  hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

  // Copy over the color table that's used by the shading
  // function for circles in the snowflake demo

  float lookupTable[COLOR_MAP_SIZE][3] = {
      {1.f, 1.f, 1.f}, {1.f, 1.f, 1.f},  {.8f, .9f, 1.f},
      {.8f, .9f, 1.f}, {.8f, 0.8f, 1.f},
  };

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable,
                     sizeof(float) * 3 * COLOR_MAP_SIZE);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height) {

  if (image)
    delete image;
  image = new Image(width, height);
}

// clearImage --
//
// Clear the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage() {

  // 256 threads per block is a healthy number
  dim3 blockDim(16, 16, 1);
  dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x,
               (image->height + blockDim.y - 1) / blockDim.y);

  kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
  hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void CudaRenderer::advanceAnimation() {
  scene->cameraRotator.rotateCamera(scene->camera);
}

void CudaRenderer::render() {
  printf("Combined Matrix: \n");
  float combinedMatrix[4][4];
  float *deviceCombinedMatrix;
  hipMalloc(&deviceCombinedMatrix, sizeof(float) * 16);
  scene->camera.calculateViewMatrix(combinedMatrix);
  // for (int i = 0; i < 4; ++i) {
  //   printf("%f, %f, %f, %f\n", combinedMatrix[i][0], combinedMatrix[i][1],
  //          combinedMatrix[i][2], combinedMatrix[i][3]);
  // }
  for (int i = 0; i < 4; i++) {
    for (int j = 0; j < 4; j++) {
      hipMemcpy(deviceCombinedMatrix + i * 4 + j, &combinedMatrix[i][j],
                 sizeof(float), hipMemcpyHostToDevice);
    }
  }
  float *projectedVertices;
  hipMalloc(&projectedVertices, sizeof(float) * 3 * numTriangles);
  dim3 blockDim(BLOCK_WIDTH * BLOCK_WIDTH);
  dim3 gridDim(
      ceil((double)numTriangles / (double)(BLOCK_WIDTH * BLOCK_WIDTH)));
  kernelProjectVertices<<<gridDim, blockDim>>>(deviceCombinedMatrix,
                                               projectedVertices);
  cudaCheckError(hipDeviceSynchronize());

  int imageWidth = image->width;
  int imageHeight = image->height;
  blockDim = dim3(BLOCK_WIDTH, BLOCK_WIDTH);
  gridDim = dim3(ceil((double)imageWidth / (double)BLOCK_WIDTH),
                 ceil((double)imageHeight / (double)BLOCK_WIDTH));
  kernelRenderPixels<<<gridDim, blockDim>>>(projectedVertices);
  cudaCheckError(hipDeviceSynchronize())
}
